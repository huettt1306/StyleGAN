#include "hip/hip_runtime.h"
#include "layer.h"
#include <omp.h>
#include <chrono>
#include <map>
#include <cmath>
#include <cstring>
#include <immintrin.h>
#include <sched.h>

class Timer {
private:
  std::chrono::time_point<std::chrono::high_resolution_clock> start;
public:
  Timer() { start = std::chrono::high_resolution_clock::now(); }
  double elapsed() {
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end - start;
    return duration.count();
  }
};

static std::map<std::string, double> time_map;
#define ITILESIZE (32)
#define JTILESIZE (256)
#define KTILESIZE (128)

/*
  * PixelNorm
  * @param [in & out] inout: [N, C]
  * Normalizes the input tensor along dim=1.
  * Equivalent to: input * torch.rsqrt(torch.mean(input ** 2, dim=1, keepdim=True) + 1e-8)
  */

void PixelNorm(Tensor *inout) {
  Timer timer;
  size_t C = inout->shape[1];
  
  float mean_squares = 0.f;
  for (size_t i = 0; i < C; i++) {
    mean_squares += inout->buf[i] * inout->buf[i];
  }
  mean_squares /= C;
  float norm_factor = rsqrtf(mean_squares + 1e-8f);

  #pragma omp parallel for simd
  for (size_t i = 0; i < C; i++) {
    inout->buf[i] *= norm_factor;
  }

  time_map["PixelNorm"] += timer.elapsed();
}

/*
 * Upsample and Pad
 * input shape = (N, C, H, W)
 * output shape = (N, C, OH, OW)
 *   where OH = H * up + pad0 + pad1,
 *         OW = W * up + pad0 + pad1
 */
void UpsamplePad(Tensor *input, Tensor *output, int up, int pad0, int pad1) {
  Timer timer;
  size_t N = input->shape[0];
  size_t C = input->shape[1]; 
  size_t H = input->shape[2];
  size_t W = input->shape[3];

  size_t OH = up * H + pad0 + pad1;
  size_t OW = up * W + pad0 + pad1;

  memset(output->buf, 0, N * C * OH * OW * sizeof(float));

  #pragma omp parallel for collapse(2)
  for (size_t c = 0; c < C; ++c) {
    for (size_t h = 0; h < H; ++h) {
      for (size_t w = 0; w < W; ++w) {
        output->buf[c * OH * OW + (h * up + pad0) * OW + w * up + pad0] +=
        input->buf[c * H * W + h * W + w];
      }
    }
  }
  time_map["UpsamplePad"] += timer.elapsed();
}

/*
 * Matrix multiplication: C = A * B
 * A: (M, K), B: (K, N), C: (M, N)
 */
void mat_mul(float *A, float *B, float *C, int M, int N, int K) {
  Timer timer;
      
  #pragma omp parallel num_threads(32)
  {
  int tid = omp_get_thread_num();
  cpu_set_t cpuset;
  CPU_ZERO(&cpuset);
  CPU_SET(tid, &cpuset);
  sched_setaffinity(0, sizeof(cpu_set_t), &cpuset);

  #pragma omp for simd
  for (int i = 0; i < M * N; i++) {
    C[i] = 0.0f;
  }

  #pragma omp for collapse(2)
  for (int i = 0; i < M; i += ITILESIZE) {
    for (int j = 0; j < N; j += JTILESIZE) {
      for (int k = 0; k < K; k += KTILESIZE) {
        for (int kk = k; kk < K && kk < k + KTILESIZE; kk+=8) {
          for (int ii = i;ii < M && ii < i + ITILESIZE; ii++) {
            __m256 a0 = _mm256_set1_ps(A[(ii+0)*K+(kk+0)]);
            __m256 a1 = _mm256_set1_ps(A[(ii+0)*K+(kk+1)]);
            __m256 a2 = _mm256_set1_ps(A[(ii+0)*K+(kk+2)]);
            __m256 a3 = _mm256_set1_ps(A[(ii+0)*K+(kk+3)]);
            __m256 a4 = _mm256_set1_ps(A[(ii+0)*K+(kk+4)]);
            __m256 a5 = _mm256_set1_ps(A[(ii+0)*K+(kk+5)]);
            __m256 a6 = _mm256_set1_ps(A[(ii+0)*K+(kk+6)]);
            __m256 a7 = _mm256_set1_ps(A[(ii+0)*K+(kk+7)]);

            for (int jj = j; jj < N && jj < j + JTILESIZE; jj+=16) {
              __m256 c0 = _mm256_load_ps(&C[(ii+0) * N + jj]);


              __m256 b0 = _mm256_load_ps(&B[(kk+0) * N + jj]);
              __m256 b1 = _mm256_load_ps(&B[(kk+1) * N + jj]);
              __m256 b2 = _mm256_load_ps(&B[(kk+2) * N + jj]);
              __m256 b3 = _mm256_load_ps(&B[(kk+3) * N + jj]);
              __m256 b4 = _mm256_load_ps(&B[(kk+4) * N + jj]);
              __m256 b5 = _mm256_load_ps(&B[(kk+5) * N + jj]);
              __m256 b6 = _mm256_load_ps(&B[(kk+6) * N + jj]);
              __m256 b7 = _mm256_load_ps(&B[(kk+7) * N + jj]);

              c0 = _mm256_fmadd_ps(a0, b0, c0);
              c0 = _mm256_fmadd_ps(a1, b1, c0);
              c0 = _mm256_fmadd_ps(a2, b2, c0);
              c0 = _mm256_fmadd_ps(a3, b3, c0);
              c0 = _mm256_fmadd_ps(a4, b4, c0);
              c0 = _mm256_fmadd_ps(a5, b5, c0);
              c0 = _mm256_fmadd_ps(a6, b6, c0);
              c0 = _mm256_fmadd_ps(a7, b7, c0);

              __m256 d0 = _mm256_load_ps(&C[(ii+0) * N + jj+8]);

              __m256 e0 = _mm256_load_ps(&B[(kk+0) * N + jj+8]);
              __m256 e1 = _mm256_load_ps(&B[(kk+1) * N + jj+8]);
              __m256 e2 = _mm256_load_ps(&B[(kk+2) * N + jj+8]);
              __m256 e3 = _mm256_load_ps(&B[(kk+3) * N + jj+8]);
              __m256 e4 = _mm256_load_ps(&B[(kk+4) * N + jj+8]);
              __m256 e5 = _mm256_load_ps(&B[(kk+5) * N + jj+8]);
              __m256 e6 = _mm256_load_ps(&B[(kk+6) * N + jj+8]);
              __m256 e7 = _mm256_load_ps(&B[(kk+7) * N + jj+8]);

              d0 = _mm256_fmadd_ps(a0, e0, d0);
              d0 = _mm256_fmadd_ps(a1, e1, d0);
              d0 = _mm256_fmadd_ps(a2, e2, d0);
              d0 = _mm256_fmadd_ps(a3, e3, d0);
              d0 = _mm256_fmadd_ps(a4, e4, d0);
              d0 = _mm256_fmadd_ps(a5, e5, d0);
              d0 = _mm256_fmadd_ps(a6, e6, d0);
              d0 = _mm256_fmadd_ps(a7, e7, d0);

              _mm256_store_ps(&C[(ii+0)*N+jj], c0);
              _mm256_store_ps(&C[(ii+0)*N+jj+8], d0);
            }
          }
        }
      }
    }
  }

  }

  time_map["matmul"] += timer.elapsed();
}

static inline size_t round_up(size_t x, size_t a) { return (x + a - 1) / a * a; }

/*
 * im2col
 * image shape = (N, C, H, W)
 * col shape = (round_up(C x R x S, 8), N x OH x OW)
 * If C x R x S is not divisible by 8, it will be padded to the next multiple of 8.
 */
void im2col (const Tensor *img, float *col, int N, int C, int H, int W, int OH, int OW,
             int R, int S, int stride, int pad, int dilation) {
  Timer timer;

  const int img_stride_c = H * W;
  const int num_patches = OH * OW;

  #pragma omp parallel for collapse(4)
  for (int n = 0; n < N; n++) {
    for (int c = 0; c < C; c++) {
      for (int r = 0; r < R; r++) {
        for (int s = 0; s < S; s++) {
          int k = c * R * S + r * S + s;
          int h_start = r * dilation - pad;
          int w_start = s * dilation - pad;

          for (int oh = 0, h = h_start; oh < OH; ++oh, h += stride) {
            for (int ow = 0, w = w_start; ow < OW; ++ow, w += stride) {
              float val = 0.0f;
              if (h >= 0 && h < H && w >= 0 && w < W) {
                val = img->buf[n * C * H * W + c * img_stride_c + h * W + w];
              }
              
              col[k * N * num_patches + n * num_patches + oh * OW + ow] = val;
            }
          }
        }
      }
    }
  }
  
  time_map["im2col"] += timer.elapsed();
}

/*
 * Add padding to weight matrix
 * A: weight matrix (K, CRS) = (M, N)
 * B: padded weight matrix (K, round_up(CRS, 8)) = (M_, N_)
*/
void addPadding(float *A, float *B, int K, int C, int R, int S) {
  Timer timer;

  int N_ = round_up(C * R * S, 8);
  int N = C * R * S;

  #pragma omp parallel for
  for (int i = 0; i < K; ++i) {
    float *dst = B + (size_t)i * N_;
    const float *src = A + (size_t)i * N;
    memcpy(dst, src, N * sizeof(float));
    if (N_ > N) {
      memset(dst + N, 0, (N_ - N) * sizeof(float));
    }
  }

  time_map["addPadding"] += timer.elapsed();
}

/*
 * Convolution
 * input shape = (N, C, H, W)
 * weight shape = (K, C, R, S)
 * bias shape = (K)
 * output shape = (N, K, OH, OW)
 *   where OH = (H + 2 * pad - dilation * (R - 1) - 1) / stride + 1,
 *         OW = (W + 2 * pad - dilation * (S - 1) - 1) / stride + 1
 */
void Conv2d(Tensor *input, Tensor *weight, Tensor *bias, Tensor *output,
            int stride, int pad, int dilation, bool has_bias) {

  Timer timer;
  int N = input->shape[0], C = input->shape[1], H = input->shape[2], W = input->shape[3];
  int K = weight->shape[0], R = weight->shape[2], S = weight->shape[3];
  int OH = output->shape[2], OW = output->shape[3];

  int _M = K;
  int _N = N * OH * OW;
  int _K = round_up(C * R * S, 8);

  float *col = (float *)aligned_alloc(32, _N * _K * sizeof(float));
  float *wei = (float *)aligned_alloc(32, _M * _K * sizeof(float));

  addPadding(weight->buf, wei, K, C, R, S);
  im2col(input, col, N, C, H, W, OH, OW, R, S, stride, pad, dilation);
  mat_mul(wei, col, output->buf, _M, _N, _K);

  if (has_bias) {
    #pragma omp parallel for simd
    for (int oh = 0; oh < OH; ++oh) {
      for (int ow = 0; ow < OW; ++ow) {
        for (int k = 0; k < K; ++k) {
          output->buf[(size_t)k * OH * OW + (size_t)oh * OW + ow] += bias->buf[k];
        }
      }
    }
  }

  time_map["Conv2d"] += timer.elapsed();
}

/*
 * Add padding and transpose
 * A: weight matrix (CKRS)
 * B: padded and transposed weight matrix (KRS, C)
*/
void addPaddingTranspose(float *A, float *B, int K, int C, int R, int S) {
  Timer timer;
  int RS = R * S;

  #pragma omp parallel for
  for (int k = 0; k < K; ++k) {
    for (int r = 0; r < R; ++r) {
      for (int s = 0; s < S; ++s) {
        int kr = (k * R + r) * S + s;
        int baseB = kr * C;
        for (int c = 0; c < C; ++c) {
          // idx A: (c * K + k) * R*S + r*S + s
          int idxA = (c * K + k) * RS + r * S + s;
          B[baseB + c] = A[idxA];
        }
      }
    }
  }

    time_map["addPaddingTranspose"] += timer.elapsed();
}


/*
 * Transposed convolution
 * input shape = (N, C, H, W)
 * weight shape = (C, K, R, S)
 * output shape = (N, K, OH, OW)
 *   where OH = (H - 1) * stride - 2 * pad + R
 *         OW = (W - 1) * stride - 2 * pad + S
 */
void ConvTranspose2d(Tensor *input, Tensor *weight, Tensor *output, 
                     int stride, int pad) {
  Timer timer;
  int N = input->shape[0], C = input->shape[1], H = input->shape[2], W = input->shape[3];
  int K = weight->shape[1], R = weight->shape[2], S = weight->shape[3];
  int OH = output->shape[2], OW = output->shape[3];

  int _M = K * R * S;
  int _N = H * W;
  int _K = C;

  float *wei = (float *)aligned_alloc(32, _M * _K * sizeof(float));
  float *temp = (float *)aligned_alloc(32, _M * _N * sizeof(float));

  addPaddingTranspose(weight->buf, wei, K, C, R, S);

  mat_mul(wei, input->buf, temp, _M, _N, _K);

  memset(output->buf, 0, N * K * OH * OW * sizeof(float));
  // KRS x HW -> K x OH x OW
  #pragma omp parallel for collapse(3)
  for (int k = 0; k < K; ++k) {
    for (int r = 0; r < R; ++r) {
      for (int s = 0; s < S; ++s) {
        int kr = k * R * S + r * S + s; 
        for (int h = 0; h < H; ++h) {
          for (int w = 0; w < W; ++w) {
            int oh = h * stride - pad + r;
            int ow = w * stride - pad + s;
            if (oh >= 0 && oh < OH && ow >= 0 && ow < OW) {
              output->buf[k * (OH * OW) + oh * OW + ow] +=
              temp[kr * (H * W) + h * W + w];
            }
          }
        }
      }
    }
  }

  time_map["ConvTranspose2d"] += timer.elapsed();
}

/* Transpose
 * input shape = (N, C, H, W)
 * output shape = (C, N, H, W)
 * Transposes the first two dimensions of the input tensor.
 */
void transpose(Tensor *input, Tensor *output) {
  Timer timer;
  size_t N = input->shape[0];
  size_t C = input->shape[1];
  size_t H = input->shape[2];
  size_t W = input->shape[3];

  #pragma omp parallel for simd collapse(2)
  for (size_t n = 0; n < N; n++) {
    for (size_t c = 0; c < C; c++) {
      for (size_t h = 0; h < H; h++) {
        for (size_t w = 0; w < W; w++) {
          size_t input_idx  = ((n * C + c) * H + h) * W + w;         // NCHW
          size_t output_idx = ((c * N + n) * H + h) * W + w;         // CNHW
          output->buf[output_idx] = input->buf[input_idx];
        }
      }
    }
  }
  time_map["transpose"] += timer.elapsed();
}

/* Linear
 * @param [in1]  in: [M, K]
 * @param [in2]   w: [N, K]
 * @param [in3]   b: [N]
 * @param [out] out: [M, N]
 */
void Linear(Tensor *in, Tensor *w, Tensor *b, Tensor *out, float lr_mul) {

  Timer timer;
  size_t M = out->shape[0];
  size_t N = out->shape[1];
  size_t K = w->shape[1];

  float scale = (1.0f / sqrtf(K)) * lr_mul;

  #pragma omp parallel for 
  for (size_t m = 0; m < M; m++) {
    for (size_t n = 0; n < N; n++) {
      out->buf[m * N + n] = 0;
      for (size_t k = 0; k < K; k++) {
        out->buf[m * N + n] += in->buf[m * K + k] * w->buf[n * K + k] * scale;
      }
      out->buf[m * N + n] += b->buf[n] * lr_mul;
    }
  }
  time_map["Linear"] += timer.elapsed();
}

/* LeakyReLU
 * @param [in & out] inout: [N]
 * 'N' is the number of elements in the tensor.
 */
void LeakyReLU(Tensor *inout) {
  Timer timer;
  size_t N = inout->num_elem();

  float negative_slope = 0.2f;
  float scale = sqrtf(2.0f);

  #pragma omp parallel for simd
  for (size_t i = 0; i < N; i++) {
    if (inout->buf[i] < 0) { inout->buf[i] *= negative_slope; }
    inout->buf[i] *= scale;
  }
  time_map["LeakyReLU"] += timer.elapsed();
}

void upfir2d(Tensor *input, Tensor *kernel, Tensor *output,
               Tensor *upsample_a, Tensor *conv_a,
               int up, int pad0, int pad1) {
  Timer timer;
  // Upsample and Pad -> Conv2d (FIR filter)
  UpsamplePad(input, upsample_a, up, pad0, pad1);

  int C = upsample_a->shape[1];
  int H = upsample_a->shape[2];
  int W = upsample_a->shape[3];
  upsample_a->reshape({C, 1, H, W});
  Conv2d(upsample_a, kernel, nullptr, output, 1, 0, 1, false);
  upsample_a->reshape({1, C, H, W});

  time_map["upfir2d"] += timer.elapsed();
}

void ModulatedConv2d(Tensor *input, Tensor *style, Tensor *modulate_weight, Tensor *modulate_bias, Tensor *conv_weight, Tensor *kernel, Tensor *output,
                     Tensor *style_a, Tensor *weight_a, Tensor *demod_a, Tensor *transpose_a, Tensor *conv_a, Tensor *upsample_a, Tensor *conv2_a,
                     bool demodulate, bool upsample, int padding, int up
) {
  Timer timer;
  size_t in_C = input->shape[1];
  size_t out_C = conv_weight->shape[0];
  size_t kernel_size = conv_weight->shape[2];

  Linear(style, modulate_weight, modulate_bias, style_a, 1.0f);

  float scale = 1 / sqrtf((float) (in_C * kernel_size * kernel_size));

  #pragma omp parallel for 
  for (size_t oc = 0; oc < out_C; oc++) {
    for (size_t ic = 0; ic < in_C; ic++) {
      for (size_t k = 0; k < kernel_size * kernel_size; k++) {
        size_t idx = oc * in_C * kernel_size * kernel_size + ic * kernel_size * kernel_size + k;
        weight_a->buf[idx] = conv_weight->buf[idx] * style_a->buf[ic] * scale;
      }
    }
  }

  if (demodulate) {
    #pragma omp parallel for
    for (size_t oc = 0; oc < out_C; oc++) {
      float sum = 0.0f;
      for (size_t ic = 0; ic < in_C; ic++) {
        for (size_t k = 0; k < kernel_size * kernel_size; k++) {
          size_t idx = oc * in_C * kernel_size * kernel_size + ic * kernel_size * kernel_size + k;
          sum += weight_a->buf[idx] * weight_a->buf[idx];
        }
      }
      demod_a->buf[oc] = 1.0f / sqrtf(sum + 1e-8f);
    }

    #pragma omp parallel for
    for (size_t oc = 0; oc < out_C; oc++) {
      for (size_t ic = 0; ic < in_C; ic++) {
        for (size_t k = 0; k < kernel_size * kernel_size; k++) {
          size_t idx = oc * in_C * kernel_size * kernel_size + ic * kernel_size * kernel_size + k;
          weight_a->buf[idx] *= demod_a->buf[oc];
        }
      }
    }
  }

  if (upsample) {
    transpose(weight_a, transpose_a);
    ConvTranspose2d(input, transpose_a, conv_a, 2, 0);
    upfir2d(conv_a, kernel, output, upsample_a, conv2_a, up, 1, 1);
  }
  else {
    Conv2d(input, weight_a, nullptr, output, 1, padding, 1, false);
  }
  time_map["ModulatedConv2d"] += timer.elapsed();
}

/* Add noise to the input tensor
 * @param [in & out] inout: [N, C, H, W]
 * @param [in] noise: [H, W]
 * Adds noise to the input tensor in-place.
 */

void addNoise(Tensor *inout, Tensor *noise) {
  Timer timer;
  size_t C = inout->shape[1];
  size_t H = inout->shape[2];
  size_t W = inout->shape[3];

  #pragma omp parallel for simd collapse(2)
  for (size_t c = 0; c < C; c++) {
    for (size_t h = 0; h < H; h++) {
      for (size_t w = 0; w < W; w++) {
        size_t idx = (c * H + h) * W + w;
        inout->buf[idx] += noise->buf[h * W + w];
      }
    }
  }
  time_map["addNoise"] += timer.elapsed();
}

/* Add bias to the input tensor
 * @param [in & out] inout: [N, C, H, W]
 * @param [in] bias: [C]
 * Adds bias to the input tensor in-place.
 */

void addBias(Tensor *inout, Tensor *bias) {
  Timer timer;
  size_t C = inout->shape[1];
  size_t H = inout->shape[2];
  size_t W = inout->shape[3];

  #pragma omp parallel for simd collapse(2) 
  for (size_t c = 0; c < C; c++) {
    for (size_t h = 0; h < H; h++) {
      for (size_t w = 0; w < W; w++) {
        size_t idx = (c * H + h) * W + w;
        inout->buf[idx] += bias->buf[c];
      }
    }
  }
  time_map["addBias"] += timer.elapsed();
}

/*
 * Element-wise addition of two tensors
 * @param [in & out] inout: [N, C, H, W]
 * @param [in] addend: [N, C, H, W]
 * Adds the elements of addend to inout in-place.
 */
void elemAdd(Tensor *inout, Tensor *addend) {
  Timer timer;
  size_t N = inout->num_elem();

  #pragma omp parallel for simd
  for (size_t i = 0; i < N; i++) {
    inout->buf[i] += addend->buf[i];
  }
  time_map["elemAdd"] += timer.elapsed();
}

void StyledConv(Tensor *input, Tensor *style, Tensor *modulate_weight, Tensor *modulate_bias, Tensor *conv_weight, Tensor *conv_bias, Tensor *kernel, Tensor *noise, Tensor *output,
                Tensor *style_a, Tensor *weight_a, Tensor *demod_a, Tensor *transpose_a, Tensor *conv_a, Tensor *upsample_a, Tensor *conv2_a,
                bool demodulate, bool upsample, int padding) {
  Timer timer;
  ModulatedConv2d(input, style, modulate_weight, modulate_bias, conv_weight, kernel, output,
                  style_a, weight_a, demod_a, transpose_a, conv_a, upsample_a, conv2_a,
                  demodulate, upsample, padding, 1);
  addNoise(output, noise);
  addBias(output, conv_bias);
  LeakyReLU(output);
  time_map["StyledConv"] += timer.elapsed();
}

void ToRGB(Tensor *input, Tensor *skip, Tensor *style, Tensor *modulate_weight, Tensor *modulate_bias, Tensor *conv_weight, Tensor *conv_bias, Tensor *kernel, Tensor *output,
           Tensor *style_a, Tensor *weight_a, Tensor *demod_a, Tensor *transpose_a, Tensor *conv_a, Tensor *upsample_a, Tensor *conv2_a, Tensor *skip_upsample_a, Tensor *skip_conv_a, Tensor *skip_a,
           bool demodulate, bool upsample, int padding) {
  Timer timer;
  ModulatedConv2d(input, style, modulate_weight, modulate_bias, conv_weight, kernel, output,
                  style_a, weight_a, demod_a, transpose_a, conv_a, upsample_a, conv2_a,
                  demodulate, upsample, padding, 2);
  addBias(output, conv_bias);

  if (skip != nullptr) {
    upfir2d(skip, kernel, skip_a, skip_upsample_a, skip_conv_a, 2, 2, 1);
    elemAdd(output, skip_a);
  }
  time_map["ToRGB"] += timer.elapsed();
}

void printTimeMap() {
  printf("\n-------------------------------------\n");
  for (const auto &entry : time_map) {
    printf("%s: %.6f seconds\n", entry.first.c_str(), entry.second);
  }
  time_map.clear();    
}