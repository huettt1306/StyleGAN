#include "hip/hip_runtime.h"
#include "layer.h"
#include <omp.h>
#include <chrono>
#include <map>
#include <cmath>
#include <cstring>

class Timer {
private:
  std::chrono::time_point<std::chrono::high_resolution_clock> start;
public:
  Timer() { start = std::chrono::high_resolution_clock::now(); }
  double elapsed() {
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end - start;
    return duration.count();
  }
};

static std::map<std::string, double> time_map;


/*
  * PixelNorm
  * @param [in & out] inout: [N, C]
  * Normalizes the input tensor along dim=1.
  * Equivalent to: input * torch.rsqrt(torch.mean(input ** 2, dim=1, keepdim=True) + 1e-8)
  */

void PixelNorm(Tensor *inout) {
  Timer timer;
  size_t C = inout->shape[1];
  
  float mean_squares = 0.f;
  for (size_t i = 0; i < C; i++) {
    mean_squares += inout->buf[i] * inout->buf[i];
  }
  mean_squares /= C;
  float norm_factor = rsqrtf(mean_squares + 1e-8f);

  for (size_t i = 0; i < C; i++) {
    inout->buf[i] *= norm_factor;
  }

  time_map["PixelNorm"] += timer.elapsed();
}

/*
 * Upsample and Pad
 * input shape = (N, C, H, W)
 * output shape = (N, C, OH, OW)
 *   where OH = H * up + pad0 + pad1,
 *         OW = W * up + pad0 + pad1
 */
void UpsamplePad(Tensor *input, Tensor *output, int up, int pad0, int pad1) {
  Timer timer;
  size_t N = input->shape[0];
  size_t C = input->shape[1]; 
  size_t H = input->shape[2];
  size_t W = input->shape[3];

  size_t OH = up * H + pad0 + pad1;
  size_t OW = up * W + pad0 + pad1;

  memset(output->buf, 0, N * C * OH * OW * sizeof(float));
  for (size_t c = 0; c < C; ++c) {
      for (size_t h = 0; h < H; ++h) {
          for (size_t w = 0; w < W; ++w) {
              output->buf[c * OH * OW + (h * up + pad0) * OW + w * up + pad0] +=
                  input->buf[c * H * W + h * W + w];
          }
      }
  }
  time_map["UpsamplePad"] += timer.elapsed();
}

/*
 * Convolution
 * input shape = (N, C, H, W)
 * weight shape = (K, C, R, S)
 * bias shape = (K)
 * output shape = (N, K, OH, OW)
 *   where OH = (H + 2 * pad - dilation * (R - 1) - 1) / stride + 1,
 *         OW = (W + 2 * pad - dilation * (S - 1) - 1) / stride + 1
 */
void Conv2d(Tensor *input, Tensor *weight, Tensor *bias, Tensor *output,
            int stride, int pad, int dilation, bool has_bias) {
  Timer timer;
  int N = input->shape[0], C = input->shape[1], H = input->shape[2], W = input->shape[3];
  int K = weight->shape[0], R = weight->shape[2], S = weight->shape[3];
  int OH = output->shape[2], OW = output->shape[3];

  #pragma omp parallel for collapse(4)
  for (int n = 0; n < N; ++n) {
    for (int k = 0; k < K; ++k) {
      for (int oh = 0; oh < OH; ++oh) {
        for (int ow = 0; ow < OW; ++ow) {
          float o = has_bias ? bias->buf[k] : 0;
          for (int c = 0; c < C; ++c) {
            int minr = max(0, (pad - oh * stride + dilation - 1) / dilation);
            int maxr = min(R, (H + pad - oh * stride + dilation - 1) / dilation);
            int mins = max(0, (pad - ow * stride + dilation - 1) / dilation);
            int maxs = min(S, (W + pad - ow * stride + dilation - 1) / dilation);
            for (int r = minr, h = oh * stride - pad + r * dilation; r < maxr; ++r, h += dilation) {
              for (int s = mins, w = ow * stride - pad + s * dilation; s < maxs; ++s, w += dilation) {
                float i = input->buf[n * C * H * W + c * H * W + h * W + w];
                float f = weight->buf[k * C * R * S + c * R * S + r * S + s];
                o += i * f;
              }
            }
          }
          output->buf[n * K * OH * OW + k * OH * OW + oh * OW + ow] = o;
        }
      }
    }
  }
  time_map["Conv2d"] += timer.elapsed();
}

/*
 * Transposed convolution
 * input shape = (N, C, H, W)
 * weight shape = (C, K, R, S)
 * output shape = (N, K, OH, OW)
 *   where OH = (H - 1) * stride - 2 * pad + R
 *         OW = (W - 1) * stride - 2 * pad + S
 */
void ConvTranspose2d(Tensor *input, Tensor *weight, Tensor *output, 
                     int stride, int pad) {
  Timer timer;
  int C = input->shape[1], H = input->shape[2], W = input->shape[3];
  int K = weight->shape[1], R = weight->shape[2], S = weight->shape[3];
  int OH = output->shape[2], OW = output->shape[3];

  #pragma omp parallel for collapse(3)
  for (int k = 0; k < K; ++k) {
    for (int oh = 0; oh < OH; ++oh) {
      for (int ow = 0; ow < OW; ++ow) {
        float o = 0.0f;
        for (int c = 0; c < C; ++c) {
          int minr = max((oh + pad) % stride, oh + pad - H * stride + stride);
          int maxr = min(oh + pad + 1, R);
          int mins = max((ow + pad) % stride, ow + pad - W * stride + stride);
          int maxs = min(ow + pad + 1, S);

          for (int r = minr, h = (oh + pad - r) / stride; r < maxr; r += stride, h -= 1) {
            for (int s = mins, w = (ow + pad - s) / stride; s < maxs; s += stride, w -= 1) {
              float i = input->buf[c * H * W + h * W + w];
              float f = weight->buf[c * K * R * S + k * R * S + r * S + s];
              o += i * f;
            }
          }
        }
        output->buf[k * OH * OW + oh * OW + ow] = o;
      }
    }
  }
  time_map["ConvTranspose2d"] += timer.elapsed();
}

/* Transpose
 * input shape = (N, C, H, W)
 * output shape = (C, N, H, W)
 * Transposes the first two dimensions of the input tensor.
 */
void transpose(Tensor *input, Tensor *output) {
  Timer timer;
  size_t N = input->shape[0];
  size_t C = input->shape[1];
  size_t H = input->shape[2];
  size_t W = input->shape[3];

  for (size_t n = 0; n < N; n++) {
    for (size_t c = 0; c < C; c++) {
      for (size_t h = 0; h < H; h++) {
        for (size_t w = 0; w < W; w++) {
          size_t input_idx  = ((n * C + c) * H + h) * W + w;         // NCHW
          size_t output_idx = ((c * N + n) * H + h) * W + w;         // CNHW
          output->buf[output_idx] = input->buf[input_idx];
        }
      }
    }
  }
  time_map["transpose"] += timer.elapsed();
}

/* Linear
 * @param [in1]  in: [M, K]
 * @param [in2]   w: [N, K]
 * @param [in3]   b: [N]
 * @param [out] out: [M, N]
 */
void Linear(Tensor *in, Tensor *w, Tensor *b, Tensor *out, float lr_mul) {

  Timer timer;
  size_t M = out->shape[0];
  size_t N = out->shape[1];
  size_t K = w->shape[1];

  float scale = (1.0f / sqrtf(K)) * lr_mul;

  for (size_t m = 0; m < M; m++) {
    for (size_t n = 0; n < N; n++) {
      out->buf[m * N + n] = 0;
      for (size_t k = 0; k < K; k++) {
        out->buf[m * N + n] += in->buf[m * K + k] * w->buf[n * K + k] * scale;
      }
      out->buf[m * N + n] += b->buf[n] * lr_mul;
    }
  }
  time_map["Linear"] += timer.elapsed();
}

/* LeakyReLU
 * @param [in & out] inout: [N]
 * 'N' is the number of elements in the tensor.
 */
void LeakyReLU(Tensor *inout) {
  Timer timer;
  size_t N = inout->num_elem();

  float negative_slope = 0.2f;
  float scale = sqrtf(2.0f);

  for (size_t i = 0; i < N; i++) {
    if (inout->buf[i] < 0) { inout->buf[i] *= negative_slope; }
    inout->buf[i] *= scale;
  }
  time_map["LeakyReLU"] += timer.elapsed();
}

void upfir2d(Tensor *input, Tensor *kernel, Tensor *output,
               Tensor *upsample_a, Tensor *conv_a,
               int up, int pad0, int pad1) {
  Timer timer;
  // Upsample and Pad -> Conv2d (FIR filter)
  UpsamplePad(input, upsample_a, up, pad0, pad1);

  int C = upsample_a->shape[1];
  int H = upsample_a->shape[2];
  int W = upsample_a->shape[3];
  upsample_a->reshape({C, 1, H, W});
  Conv2d(upsample_a, kernel, nullptr, output, 1, 0, 1, false);
  upsample_a->reshape({1, C, H, W});

  time_map["upfir2d"] += timer.elapsed();
}

void ModulatedConv2d(Tensor *input, Tensor *style, Tensor *modulate_weight, Tensor *modulate_bias, Tensor *conv_weight, Tensor *kernel, Tensor *output,
                     Tensor *style_a, Tensor *weight_a, Tensor *demod_a, Tensor *transpose_a, Tensor *conv_a, Tensor *upsample_a, Tensor *conv2_a,
                     bool demodulate, bool upsample, int padding, int up
) {
  Timer timer;
  size_t in_C = input->shape[1];
  size_t out_C = conv_weight->shape[0];
  size_t kernel_size = conv_weight->shape[2];

  Linear(style, modulate_weight, modulate_bias, style_a, 1.0f);

  float scale = 1 / sqrtf((float) (in_C * kernel_size * kernel_size));

  for (size_t oc = 0; oc < out_C; oc++) {
    for (size_t ic = 0; ic < in_C; ic++) {
      for (size_t k = 0; k < kernel_size * kernel_size; k++) {
        size_t idx = oc * in_C * kernel_size * kernel_size + ic * kernel_size * kernel_size + k;
        weight_a->buf[idx] = conv_weight->buf[idx] * style_a->buf[ic] * scale;
      }
    }
  }

  if (demodulate) {
    for (size_t oc = 0; oc < out_C; oc++) {
      float sum = 0.0f;
      for (size_t ic = 0; ic < in_C; ic++) {
        for (size_t k = 0; k < kernel_size * kernel_size; k++) {
          size_t idx = oc * in_C * kernel_size * kernel_size + ic * kernel_size * kernel_size + k;
          sum += weight_a->buf[idx] * weight_a->buf[idx];
        }
      }
      demod_a->buf[oc] = 1.0f / sqrtf(sum + 1e-8f);
    }

    for (size_t oc = 0; oc < out_C; oc++) {
      for (size_t ic = 0; ic < in_C; ic++) {
        for (size_t k = 0; k < kernel_size * kernel_size; k++) {
          size_t idx = oc * in_C * kernel_size * kernel_size + ic * kernel_size * kernel_size + k;
          weight_a->buf[idx] *= demod_a->buf[oc];
        }
      }
    }
  }

  if (upsample) {
    transpose(weight_a, transpose_a);
    ConvTranspose2d(input, transpose_a, conv_a, 2, 0);
    upfir2d(conv_a, kernel, output, upsample_a, conv2_a, up, 1, 1);
  }
  else {
    Conv2d(input, weight_a, nullptr, output, 1, padding, 1, false);
  }
  time_map["ModulatedConv2d"] += timer.elapsed();
}

/* Add noise to the input tensor
 * @param [in & out] inout: [N, C, H, W]
 * @param [in] noise: [H, W]
 * Adds noise to the input tensor in-place.
 */

void addNoise(Tensor *inout, Tensor *noise) {
  Timer timer;
  size_t C = inout->shape[1];
  size_t H = inout->shape[2];
  size_t W = inout->shape[3];

  for (size_t c = 0; c < C; c++) {
    for (size_t h = 0; h < H; h++) {
      for (size_t w = 0; w < W; w++) {
        size_t idx = (c * H + h) * W + w;
        inout->buf[idx] += noise->buf[h * W + w];
      }
    }
  }
  time_map["addNoise"] += timer.elapsed();
}

/* Add bias to the input tensor
 * @param [in & out] inout: [N, C, H, W]
 * @param [in] bias: [C]
 * Adds bias to the input tensor in-place.
 */

void addBias(Tensor *inout, Tensor *bias) {
  Timer timer;
  size_t C = inout->shape[1];
  size_t H = inout->shape[2];
  size_t W = inout->shape[3];

  for (size_t c = 0; c < C; c++) {
    for (size_t h = 0; h < H; h++) {
      for (size_t w = 0; w < W; w++) {
        size_t idx = (c * H + h) * W + w;
        inout->buf[idx] += bias->buf[c];
      }
    }
  }
  time_map["addBias"] += timer.elapsed();
}

/*
 * Element-wise addition of two tensors
 * @param [in & out] inout: [N, C, H, W]
 * @param [in] addend: [N, C, H, W]
 * Adds the elements of addend to inout in-place.
 */
void elemAdd(Tensor *inout, Tensor *addend) {
  Timer timer;
  size_t N = inout->num_elem();

  for (size_t i = 0; i < N; i++) {
    inout->buf[i] += addend->buf[i];
  }
  time_map["elemAdd"] += timer.elapsed();
}

void StyledConv(Tensor *input, Tensor *style, Tensor *modulate_weight, Tensor *modulate_bias, Tensor *conv_weight, Tensor *conv_bias, Tensor *kernel, Tensor *noise, Tensor *output,
                Tensor *style_a, Tensor *weight_a, Tensor *demod_a, Tensor *transpose_a, Tensor *conv_a, Tensor *upsample_a, Tensor *conv2_a,
                bool demodulate, bool upsample, int padding) {
  Timer timer;
  ModulatedConv2d(input, style, modulate_weight, modulate_bias, conv_weight, kernel, output,
                  style_a, weight_a, demod_a, transpose_a, conv_a, upsample_a, conv2_a,
                  demodulate, upsample, padding, 1);
  addNoise(output, noise);
  addBias(output, conv_bias);
  LeakyReLU(output);
  time_map["StyledConv"] += timer.elapsed();
}

void ToRGB(Tensor *input, Tensor *skip, Tensor *style, Tensor *modulate_weight, Tensor *modulate_bias, Tensor *conv_weight, Tensor *conv_bias, Tensor *kernel, Tensor *output,
           Tensor *style_a, Tensor *weight_a, Tensor *demod_a, Tensor *transpose_a, Tensor *conv_a, Tensor *upsample_a, Tensor *conv2_a, Tensor *skip_upsample_a, Tensor *skip_conv_a, Tensor *skip_a,
           bool demodulate, bool upsample, int padding) {
  Timer timer;
  ModulatedConv2d(input, style, modulate_weight, modulate_bias, conv_weight, kernel, output,
                  style_a, weight_a, demod_a, transpose_a, conv_a, upsample_a, conv2_a,
                  demodulate, upsample, padding, 2);
  addBias(output, conv_bias);

  if (skip != nullptr) {
    upfir2d(skip, kernel, skip_a, skip_upsample_a, skip_conv_a, 2, 2, 1);
    elemAdd(output, skip_a);
  }
  time_map["ToRGB"] += timer.elapsed();
}

void printTimeMap() {
  printf("\n-------------------------------------\n");
  for (const auto &entry : time_map) {
    printf("%s: %.6f seconds\n", entry.first.c_str(), entry.second);
  }
  time_map.clear();
}